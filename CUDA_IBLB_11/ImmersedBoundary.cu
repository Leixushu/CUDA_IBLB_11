#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstdio>

#include "hip/hip_runtime.h"
#include ""
#include "ImmersedBoundary.cuh"
#include "hip/device_functions.h"

#define PI 3.14159
//__device__ const double RHO_0 = 1.;
//__device__ const double C_S = 0.57735;

__constant__ double c_l[9 * 2] =		//VELOCITY COMPONENTS
{
	0.,0. ,
	1.,0. , 0.,1. , -1.,0. , 0.,-1. ,
	1.,1. , -1.,1. , -1.,-1. , 1.,-1.
};

__device__ double sqroot(double square)
{
	const double MINDIFF = 0.1;
	double root = square / 0.75, last, diff = 1;
	if (square <= 0) return 0;
	do {
		last = root;
		root = (root + square / root) / 2;
		diff = root - last;
	} while (diff > MINDIFF || diff < -MINDIFF);
	return root;
}

__device__ double delta(const double & xs, const double & ys, const int & x, const int & y)
{
	double deltax(0.), deltay(0.), delta(0.);

	double dx = abs(x - xs);
	double dy = abs(y - ys);

	double a(0.), b(0.), d(0.);
	int c(0);

	if (dx <= 1.5)
	{
		if (dx <= 0.5)
		{
			//deltax = (1. / 3.)*(1. + sqrt(-3. * dx*dx + 1.));
			a = 0.33333;
			b = 1.;
			c = 1;
			d = dx;
		}
		else //deltax = (1. / 6.)*(5. - 3. * dx - sqrt(-3. * (1. - dx)*(1. - dx) + 1.));
		{
			a = 0.16667;
			b = 5.-3.*dx;
			c = -1;
			d = 1-dx;
		}
	}

	deltax = a*(b + c*sqrt(-3.*d*d + 1));

	a = 0.;
	b = 0.;
	c = 0;
	d = 0.;

	if (dy <= 1.5)
	{
		if (dy <= 0.5)
		{
			//deltay = (1. / 3.)*(1. + sqrt(-3. * dy*dy + 1.));
			a = 0.33333;
			b = 1.;
			c = 1;
			d = dy;
		}
		else //deltay = (1. / 6.)*(5. - 3. * dy - sqrt(-3. * (1. - dy)*(1. - dy) + 1.));
		{
			a = 0.16667;
			b = 5. - 3.*dy;
			c = -1;
			d = 1 - dy;
		}
	}

	deltay = a*(b + c*sqrt(-3.*d*d + 1));

	delta = deltax * deltay;

	return delta;
}

__device__ void DoubleAtomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do
	{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
}

__global__ void interpolate(const double * rho, const double * u, const int Ns, const double * u_s, double * F_s, const double * s, const int XDIM)
{

	int i(0), j(0), k(0), x0(0), y0(0), x(0), y(0);

	double xs(0.), ys(0.), del(0.);


	k = blockIdx.x*blockDim.x + threadIdx.x;


	{
		F_s[2 * k + 0] = 0.;
		F_s[2 * k + 1] = 0.;

		xs = s[k * 2 + 0];
		ys = s[k * 2 + 1];

		x0 = nearbyint(xs);
		y0 = nearbyint(ys);

		for (i = 0; i < 9; i++)
		{
			x = nearbyint(x0 + c_l[i * 2 + 0]);
			y = nearbyint(y0 + c_l[i * 2 + 1]);

			j = y*XDIM + x;

			del = delta(xs, ys, x, y);

			F_s[2 * k + 0] += 2.*(1. * 1. * del)*rho[j] * (u_s[2 * k + 0] - u[2 * j + 0]);
			F_s[2 * k + 1] += 2.*(1. * 1. * del)*rho[j] * (u_s[2 * k + 1] - u[2 * j + 1]);
		}

	}

	__syncthreads();
}

// rho[SIZE]: fluid density	u[2*size]: fluid velocity	f[9*size]: density function		Ns: No. of cilia boundary points	u_s[2*Ns]: cilia velocity	F_s[2*Ns]: cilia force	
// force[2*size]: fluid force	s[2*Ns]: cilia position	XDIM: x dimension	Q: Net flow		epsilon[Ns]: boundary point switching

__global__ void spread(const double * rho, double * u, const double * f, const int Ns, const double * u_s, const double * F_s, double * force, const double * s, const int XDIM, double * Q, const int * epsilon)
{
	int j(0), k(0), x(0), y(0);

	int n(0), m(0);

	double xs(0.), ys(0.), del(0.);

	int size = 192 * XDIM;

	////////////////////////////////////////////////////////////////START//////////////////////////////////////////////////

	const int tile = 128;	//size of a tile, same as blockdim.x

	const int tpoints = tile / 2;

	int numtiles = (2 * Ns - (2 * Ns) % tile) / (tile);	//number of full tiles to populate the whole array of values
	
	int excess = (2 * Ns) % tile;	//number of values outside of full tiles
	
	__shared__ double sh_s[tile];	//shared version of s array
	__shared__ double sh_F_s[tile];	//shared version of F_s array
	__shared__ int sh_epsilon[tile];

	j = blockIdx.x*blockDim.x + threadIdx.x;	//unique thread ID

	n = threadIdx.x;		//thread ID within block

	force[0 * size + j] = 0.;		//initialise
	force[1 * size + j] = 0.;

	sh_s[n] = 0.;
	sh_F_s[n] = 0.;

	x = j%XDIM;
	y = (j - j%XDIM) / XDIM;

	double temp;

	for (m = 0; m < numtiles; m++)		//iterate for each tile within the arrays
	{
		__syncthreads();

		sh_s[n] = s[m*tile + n];		//take values from next tile in the arrays to shared memory
		sh_F_s[n] = F_s[m*tile + n];
		if(n<tpoints) sh_epsilon[n] = epsilon[m*tpoints + n];

		__syncthreads();


		for (k = 0; k < tpoints; k++)	//iterate for each value within a tile ("tile" values reporesent "tile/2" points with x and y coordinates)
		{
			xs = sh_s[2 * k + 0];		//x value
			ys = sh_s[2 * k + 1];		//y value

			del = delta(xs, ys, x, y);

			force[0 * size + j] += sh_F_s[2 * k + 0] * del * 1. * sh_epsilon[k];		//calculate force x
			force[1 * size + j] += sh_F_s[2 * k + 1] * del * 1. * sh_epsilon[k];		//calculate force y

			//__syncthreads();
		}

		__syncthreads();
	}

	

	if (n < excess)		//if there are excess values after the arrays have been split into tiles, and only execute for that many threads
	{
		sh_s[n] = s[numtiles*tile + n];		//take values from excess into shared memory
		sh_F_s[n] = F_s[numtiles*tile + n];
	}
	else
	{
		sh_s[n] = -100.;		//dummy values
		sh_F_s[n] = 0.;
	}
	
	__syncthreads();

		for (k = 0; k < tpoints; k++)	//iterate for all remaining values
		{
			xs = sh_s[k * 2 + 0];		//x value
			ys = sh_s[k * 2 + 1];		//y value

			del = delta(xs, ys, x, y);

			force[0 * size + j] += sh_F_s[2 * k + 0] * del * 1.*epsilon[numtiles*tpoints + k];		//calculate force x
			force[1 * size + j] += sh_F_s[2 * k + 1] * del * 1.*epsilon[numtiles*tpoints + k];		//calculate force y

			//__syncthreads();
		}
		
	__syncthreads();

	//this is the original code, without using shared memory
	/*for (k = 0; k < Ns; k++)
	{
		xs = s[k * 2 + 0];
		ys = s[k * 2 + 1];

		del = delta(xs, ys, x, y);

		force[0 * size + j] += F_s[2 * k + 0] * del * 1.*epsilon[k];
		force[1 * size + j] += F_s[2 * k + 1] * del * 1.*epsilon[k];
	}*/

	/////////////////////////////////////////////////////////////////END////////////////////////////////////////////////////////////

	u[2 * j + 0] = (c_l[0 * 2 + 0] * f[9 * j + 0] + c_l[1 * 2 + 0] * f[9 * j + 1] + c_l[2 * 2 + 0] * f[9 * j + 2] + 
			c_l[3 * 2 + 0] * f[9 * j + 3] + c_l[4 * 2 + 0] * f[9 * j + 4] + c_l[5 * 2 + 0] * f[9 * j + 5] + 
			c_l[6 * 2 + 0] * f[9 * j + 6] + c_l[7 * 2 + 0] * f[9 * j + 7] + c_l[8 * 2 + 0] * f[9 * j + 8] + 0.5*force[0 * size + j]) / rho[j];

	u[2 * j + 1] = (c_l[1 * 2 + 1] * f[9 * j + 1] + c_l[1 * 2 + 1] * f[9 * j + 1] + c_l[2 * 2 + 1] * f[9 * j + 2] +
			c_l[3 * 2 + 1] * f[9 * j + 3] + c_l[4 * 2 + 1] * f[9 * j + 4] + c_l[5 * 2 + 1] * f[9 * j + 5] +
			c_l[6 * 2 + 1] * f[9 * j + 6] + c_l[7 * 2 + 1] * f[9 * j + 7] + c_l[8 * 2 + 1] * f[9 * j + 8] + 0.5*force[1 * size + j]) / rho[j];

	__syncthreads();

	if (x == XDIM - 5)
	{
		//Q[0] += u[2 * j + 0]/192.;
		temp = u[2 * j + 0] / 192.;
		DoubleAtomicAdd(Q, temp);
	}

	__syncthreads();
}


