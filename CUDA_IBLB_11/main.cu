#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <ctime>
#include <sstream>

#include "hip/hip_runtime.h"
#include ""

#include "LatticeBoltzmann.cuh"
#include "ImmersedBoundary.cuh"

#include "seconds.h"



using namespace std;

//------------------------------------------PHYSICAL CONSTANTS----------------------------

#define C_S 0.577		//SPEED OF SOUND ON LATTICE
#define RHO_0 1.		//FLUID DENSITY
#define PI 3.14159		//PI

//-------------------------------------------PARAMETER SCALING----------------------------

double l_0 = 0.000006;					//6 MICRON CILIUM LENGTH
double t_0 = 0.067;						//67ms BEAT PERIOD AT 15Hz


__global__ void define_filament(const int m, const int T, const int it, const double offset, double * s, double * lasts)
{
	int n(0);

	double arcl(0.);

	double a_n[2 * 7];
	double b_n[2 * 7];

	int threadnum = blockDim.x*blockIdx.x + threadIdx.x;

	int k = threadnum;

	double A_mn[7 * 2 * 3] =
	{
		-0.449,	 0.130, -0.169,	 0.063, -0.050, -0.040, -0.068,
		2.076, -0.003,	 0.054,	 0.007,	 0.026,	 0.022,	 0.010,
		-0.072, -1.502,	 0.260, -0.123,	 0.011, -0.009,	 0.196,
		-1.074, -0.230, -0.305, -0.180, -0.069,	 0.001, -0.080,
		0.658,	 0.793, -0.251,	 0.049,	 0.009,	 0.023, -0.111,
		0.381,	 0.331,	 0.193,	 0.082,	 0.029,	 0.002,	 0.048
	};

	double B_mn[7 * 2 * 3] =
	{
		0.0, -0.030, -0.093,  0.037,  0.062,  0.016, -0.065,
		0.0,  0.080, -0.044, -0.017,  0.052,  0.007,  0.051,
		0.0,  1.285, -0.036, -0.244, -0.093, -0.137,  0.095,
		0.0, -0.298,  0.513,  0.004, -0.222,  0.035, -0.128,
		0.0, -1.034,  0.050,  0.143,  0.043,  0.098, -0.054,
		0.0,  0.210, -0.367,  0.009,  0.120, -0.024,  0.102
	};

	{
		arcl = 1.*k / 10000;

		for (n = 0; n < 7; n++)
		{
			a_n[2 * n + 0] = 0.;
			b_n[2 * n + 0] = 0.;

			a_n[2 * n + 0] += A_mn[n + 14 * 0 + 7 * 0] * pow(arcl, 0 + 1);
			b_n[2 * n + 0] += B_mn[n + 14 * 0 + 7 * 0] * pow(arcl, 0 + 1);

			a_n[2 * n + 0] += A_mn[n + 14 * 1 + 7 * 0] * pow(arcl, 1 + 1);
			b_n[2 * n + 0] += B_mn[n + 14 * 1 + 7 * 0] * pow(arcl, 1 + 1);

			a_n[2 * n + 0] += A_mn[n + 14 * 2 + 7 * 0] * pow(arcl, 2 + 1);
			b_n[2 * n + 0] += B_mn[n + 14 * 2 + 7 * 0] * pow(arcl, 2 + 1);

			a_n[2 * n + 1] = 0.;
			b_n[2 * n + 1] = 0.;

			a_n[2 * n + 1] += A_mn[n + 14 * 0 + 7 * 1] * pow(arcl, 0 + 1);
			b_n[2 * n + 1] += B_mn[n + 14 * 0 + 7 * 1] * pow(arcl, 0 + 1);

			a_n[2 * n + 1] += A_mn[n + 14 * 1 + 7 * 1] * pow(arcl, 1 + 1);
			b_n[2 * n + 1] += B_mn[n + 14 * 1 + 7 * 1] * pow(arcl, 1 + 1);

			a_n[2 * n + 1] += A_mn[n + 14 * 2 + 7 * 1] * pow(arcl, 2 + 1);
			b_n[2 * n + 1] += B_mn[n + 14 * 2 + 7 * 1] * pow(arcl, 2 + 1);

		}

		s[5 * (k + m * 10000) + 0] = 1. * 115 * a_n[2 * 0 + 0] * 0.5 + offset;
		s[5 * (k + m * 10000) + 1] = 1. * 115 * a_n[2 * 0 + 1] * 0.5;
		s[5 * (k + m * 10000) + 2] = 115 * arcl;

		for (n = 1; n < 7; n++)
		{
			s[5 * (k + m * 10000) + 0] += 1. * 115 * (a_n[2 * n + 0] * cos(n*2.*PI*it / T) + b_n[2 * n + 0] * sin(n*2.*PI*it / T));
			s[5 * (k + m * 10000) + 1] += 1. * 115 * (a_n[2 * n + 1] * cos(n*2.*PI*it / T) + b_n[2 * n + 1] * sin(n*2.*PI*it / T));
		}

		if (it > 0)
		{
			s[5 * (k + m * 10000) + 3] = s[5 * (k + m * 10000) + 0] - lasts[2 * (k + m * 10000) + 0];
			s[5 * (k + m * 10000) + 4] = s[5 * (k + m * 10000) + 1] - lasts[2 * (k + m * 10000) + 1];
		}
		

		lasts[2 * (k + m * 10000) + 0] = s[5 * (k + m * 10000) + 0];
		lasts[2 * (k + m * 10000) + 1] = s[5 * (k + m * 10000) + 1];
	}
}

__global__ void define_boundary(const int m, const int c_num, const double * boundary, double * b_points)
{
	int j(0), k(0);
	double b_length(0.);
	double step(1.);

	int threadnum = blockDim.x*blockIdx.x + threadIdx.x;

	k = threadnum;

	if (k == 0)
	{
		b_points[5 * (k + m * 100) + 0] = boundary[5 * (1 + m * 10000) + 0];
		b_points[5 * (k + m * 100) + 1] = boundary[5 * (1 + m * 10000) + 1];

		b_points[5 * (k + m * 100) + 2] = boundary[5 * (1 + m * 10000) + 3];
		b_points[5 * (k + m * 100) + 3] = boundary[5 * (1 + m * 10000) + 4];
	}
	else
	{
		b_length = k*step;

		for (j = (1 + m * 10000); j < c_num*10000; j++)
		{
			if (abs(boundary[5 * j + 2] - b_length) < 0.01)
			{
				b_points[5 * (k + m * 100) + 0] = boundary[5 * j + 0];
				b_points[5 * (k + m * 100) + 1] = boundary[5 * j + 1];

				b_points[5 * (k + m * 100) + 2] = boundary[5 * j + 3];
				b_points[5 * (k + m * 100) + 3] = boundary[5 * j + 4];

				j = c_num*10000;
			}
			else
			{
				b_points[5 * (k + m * 100) + 0] = 0.;
				b_points[5 * (k + m * 100) + 1] = 250.;

				b_points[5 * (k + m * 100) + 2] = 0.1;
				b_points[5 * (k + m * 100) + 3] = 0.1;
			}
		}
	}
}

void boundary_check(const int m, const double c_space, const int c_num, const int L, const double * s, int * epsilon)
{
	int r(0), k(0), l(0);

	int b_cross = 0;
	int lowest = 0;

	bool xclose = 0;
	bool yclose = 0;

	int r_max = 2 * L / c_space;

	double x_m(0.), y_m(0.), x_l(0.), y_l(0.);

	for (r = 1; r <= r_max; r++)
	{
		

		b_cross = 2 * L - r*c_space;

		if (b_cross > L) lowest = 0;
		else lowest = L - b_cross;

		for (k = lowest; k < L; k++)
		{
			x_m = s[2 * (k + m * 100) + 0];
			y_m = s[2 * (k + m * 100) + 1];

			for (l = lowest; l < L; l++)
			{
				xclose = 0;
				yclose = 0;

				if (m-r < 0)
				{
					x_l = s[2 * (l + (m - r + c_num) * 100) + 0];
					y_l = s[2 * (l + (m - r + c_num) * 100) + 1];
				}
				else
				{
					x_l = s[2 * (l + (m - r) * 100) + 0];
					y_l = s[2 * (l + (m - r) * 100) + 1];
				}

				if (abs(x_l - x_m) < 1) xclose = 1;

				if (abs(y_l - y_m) < 1) yclose = 1;

				if (xclose && yclose) epsilon[(k + m * 100)] = 0;

			}
		}
	}

}


int main(int argc, char * argv[])
{
	//----------------------------INITIALISING----------------------------

	unsigned int c_fraction = 1;
	unsigned int c_num = 6;
	double Re = 1.0;
	unsigned int XDIM = 300;
	unsigned int YDIM = 200;
	unsigned int T = 100000;
	unsigned int ITERATIONS = T;
	unsigned int INTERVAL = 100;
	unsigned int LENGTH = 100;
	bool ShARC = 0;
	bool BigData = 0;
	

	stringstream arg;

	arg << argv[1] << ' ' << argv[2] << ' ' << argv[3] << ' ' << argv[4] << ' ' << argv[5] << ' ' << argv[6] << ' ' << argv[7] << ' ' << argv[8];

	arg >> c_fraction >> c_num >> Re >> T >> ITERATIONS >> INTERVAL >> ShARC >> BigData;


	unsigned int c_space = LENGTH / 2;
	XDIM = c_num*c_space;
	const double centre[2] = { XDIM / 2., 0. };

	double dx = 1. / LENGTH;
	double dt = 1. / (T);
	double  SPEED = 0.8*1000/T;

	const double TAU = (SPEED*LENGTH) / (Re*C_S*C_S) + 1. / 2.;
	const double TAU2 = 1. / (12.*(TAU - (1. / 2.))) + (1. / 2.);

	time_t rawtime;
	struct tm * timeinfo;
	time(&rawtime);
	timeinfo = localtime(&rawtime);

	cout << asctime(timeinfo) << endl;

	cout << "Initialising...\n";

	unsigned int i(0), j(0), k(0), n(0), m(0);

	unsigned int it(0);
	int phase(0);
	int p_step = T * c_fraction / c_num;

	
	double offset = 0.;

	double * lasts;
	lasts = new double[2 * c_num * 10000];

	double * boundary;
	boundary = new double[5 * c_num * 10000];

	int Np = 100 * c_num;
	double * b_points;

	b_points = new double[5 * Np];

	
	const int size = XDIM*YDIM;

	for (k = 0; k < c_num*10000; k++)
	{
		boundary[5 * k + 0] = 0.;
		boundary[5 * k + 1] = 0.;
		boundary[5 * k + 2] = 0.;
		boundary[5 * k + 3] = 0.;
		boundary[5 * k + 4] = 0.;

		lasts[2 * k + 0] = 0.;
		lasts[2 * k + 1] = 0.;

	}

	

	//-------------------------------CUDA PARAMETERS DEFINITION-----------------------


	int blocksize = 500;

	int gridsize = size / blocksize;

	int blocksize2 = c_num*LENGTH;

	int gridsize2 = 1;

	if (blocksize2 > 1000)
	{
		for (blocksize2 = 1000; blocksize2 > 0; blocksize2 -= LENGTH)
		{
			if ((c_num*LENGTH) % blocksize2 == 0)
			{
				gridsize2 = (c_num*LENGTH) / blocksize2;
				break;
			}
		}
	}

	hipError_t cudaStatus;

	double Q = 0.;
	double E = 0.;

	if(ShARC) cudaStatus = hipSetDevice(3);
	else cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Failed to set CUDA device.\n");
	}

	//------------------------------------------ERROR------------------------------------------------


	double l_error = (l_0*dx)*(l_0*dx);
	double t_error = (t_0*dt)*(t_0*dt);
	double c_error = (t_0*dt)*(t_0*dt) / ((l_0*dx)*(l_0*dx));
	double Ma = 1.*SPEED / C_S;
	time_t p_runtime;


	//-------------------------------------------ASSIGN CELL VALUES ON HEAP-----------------------------

	double * u;								//VELOCITY VECTOR

	u = new double[2 * size];

	double * rho;							//DENSITY

	rho = new double[size];

	double * f0;							//EQUILIBRIUM DISTRIBUTION FUNCTION

	f0 = new double[9 * size];

	double * f;								//DISTRIBUTION FUNCTION

	f = new double[9 * size];

	double * f1;							//POST COLLISION DISTRIBUTION FUNCTION

	f1 = new double[9 * size];

	double * force;							//MACROSCOPIC BODY FORCE VECTOR

	force = new double[2 * size];

	double * F;								//LATTICE BOLTZMANN FORCE

	F = new double[9 * size];

	unsigned int Ns = LENGTH * c_num;		//NUMBER OF BOUNDARY POINTS


	double * s;							//BOUNDARY POINTS

	double * u_s;						//BOUNDARY POINT VELOCITY

	double * F_s;						//BOUNDARY FORCE

	int * epsilon;

	s = new double[2 * Ns];

	u_s = new double[2 * Ns];

	F_s = new double[2 * Ns];

	epsilon = new int[Ns];

	for (k = 0; k < Ns; k++)
	{
		epsilon[k] = 1;
	}


	//----------------------------------------CREATE DEVICE VARIABLES-----------------------------

	double * d_u;								//VELOCITY VECTOR

	double * d_rho;							//DENSITY

	double * d_f0;							//EQUILIBRIUM DISTRIBUTION FUNCTION

	double * d_f;								//DISTRIBUTION FUNCTION

	double * d_f1;							//POST COLLISION DISTRIBUTION FUNCTION

	double * d_centre;

	double * d_force;

	double * d_F;

	double * d_F_s;

	double * d_s;

	double * d_u_s;

	int * d_epsilon;

	double * d_Q;

	

	double * d_lasts;

	double * d_boundary;

	double * d_b_points;



	//---------------------------CUDA MALLOC-------------------------------------------------------------
	{
		cudaStatus = hipMalloc((void**)&d_u, 2 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_rho, size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_f0, 9 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_f, 9 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_f1, 9 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed,");
		}

		cudaStatus = hipMalloc((void**)&d_centre, 2 * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_force, 2 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_F, 9 * size * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

		cudaStatus = hipMalloc((void**)&d_Q, sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
		}

	}

	{

		cudaStatus = hipMalloc((void**)&d_F_s, 2 * Ns * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of F_s failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_s, 2 * Ns * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of s failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_u_s, 2 * Ns * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of u_s failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_epsilon, Ns * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of epsilon failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_lasts, 2 * c_num * 10000 * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of u_s failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_boundary, 5 * c_num * 10000 * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of u_s failed!\n");
		}

		cudaStatus = hipMalloc((void**)&d_b_points, 5 * Np * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc of u_s failed!\n");
		}

	}

	//----------------------------------------DEFINE DIRECTORIES----------------------------------
	
	string output_data = "Data/Test/";

	if(ShARC) output_data = "/shared/soft_matter_physics2/User/Phq16ja/ShARC_Data/";
	else output_data = "C:/Users/phq16ja/Documents/Data/";
		//output_data = "//uosfstore.shef.ac.uk/shared/soft_matter_physics2/User/Phq16ja/Local_Data/";

	string raw_data = output_data + "Raw/";
	raw_data += to_string(c_num);
	raw_data += "/";

	string cilia_data =  output_data + "Cilia/";
	cilia_data += to_string(c_num);
	cilia_data += "/";

	string img_data = output_data + "Img/";
	img_data += to_string(c_num);
	img_data += "/";
	
	img_data += to_string(c_num);

	string outfile = cilia_data;

	//----------------------------------------BOUNDARY INITIALISATION------------------------------------------------

	string flux = output_data + "/Flux/" + to_string(c_fraction) + "_" + to_string(c_num) +"-flux.dat";

	string parameters = raw_data + "/SimLog.txt";

	string input = "Data/cilium/";
	input += to_string(c_num);
	input += "/";

	ofstream fsA(input.c_str());

	ofstream fsB(flux.c_str());

	ofstream fsC(parameters.c_str());

	fsB.open(flux.c_str(), ofstream::trunc);

	fsB.close();

	fsC.open(parameters.c_str(), ofstream::trunc);

	fsC.close();


	//----------------------------------------INITIALISE ALL CELL VALUES---------------------------------------

	for (j = 0; j < XDIM*YDIM; j++)
	{
		rho[j] = RHO_0;
		u[2 * j + 0] = 0.0;
		u[2 * j + 1] = 0.0;

		force[2 * j + 0] = 0.;
		force[2 * j + 1] = 0.;


		for (i = 0; i < 9; i++)
		{
			f0[9 * j + i] = 0.;
			f[9 * j + i] = 0.;
			f1[9 * j + i] = 0.;
			F[9 * j + i] = 0.;
		}

	}

	//------------------------------------------------------COPY INITIAL VALUES TO DEVICE-----------------------------------------------------------

	//CUDA MEMORY COPIES
	{
		cudaStatus = hipMemcpy(d_u, u, 2 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_rho, rho, size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_f0, f0, 9 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_f, f, 9 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_f1, f1, 9 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_centre, centre, 2 * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_force, force, 2 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_F, F, 9 * size * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(d_lasts, lasts, 2 * c_num * 10000 * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy of lasts failed!"); }

		cudaStatus = hipMemcpy(d_boundary, boundary, 5 * c_num * 10000 * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy of boundary failed!"); }


		cudaStatus = hipMemcpy(d_Q, &Q, sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}


	}

	//------------------------------------------------------SET INITIAL DISTRIBUTION TO EQUILIBRIUM-------------------------------------------------

	equilibrium << <gridsize, blocksize >> > (d_u, d_rho, d_f0, d_force, d_F, XDIM, YDIM, TAU);				//INITIAL EQUILIBRIUM SET

	{																										// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "first equilibrium launch failed: %s\n", hipGetErrorString(cudaStatus));
		}

		cudaStatus = hipMemcpy(f0, d_f0, 9 * size * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		cudaStatus = hipMemcpy(F, d_F, 9 * size * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}


	}

	for (j = 0; j < XDIM*YDIM; j++)
	{
		for (i = 0; i < 9; i++)
		{
			f[9 * j + i] = f0[9 * j + i];
		}
	}

	cudaStatus = hipMemcpy(d_f, f, 9 * size * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of f failed!\n");
	}



	//-----------------------------------------------------OUTPUT PARAMETERS------------------------------------------------------------------------


	fsC.open(parameters.c_str(), ofstream::trunc);

	fsC.close();

	fsC.open(parameters.c_str(), ofstream::app);

	fsC << asctime(timeinfo) << endl;
	fsC << "Size: " << XDIM << "x" << YDIM << endl;
	fsC << "Iterations: " << ITERATIONS << endl;
	fsC << "Reynolds Number: " << Re << endl;
	fsC << "Relaxation times: " << TAU << ", " << TAU2 << endl;
	//if (TAU <= 0.6) fsC << "POSSIBLE INSTABILITY! Relaxation time: " << TAU << endl;
	//if (TAU >= 2.01) fsC << "POSSIBLE INACCURACY! Relaxation time: " << TAU << endl;

	fsC << "Spatial step: " << dx*l_0 << "m" << endl;
	fsC << "Time step: " << dt*t_0 << "s" << endl;
	fsC << "Mach number: " << Ma << endl;
	fsC << "Spatial discretisation error: " << l_error << endl;
	fsC << "Time discretisation error: " << t_error << endl;
	fsC << "Compressibility error: " << c_error << endl;


	fsC << "\nThreads per block: " << blocksize << endl;
	fsC << "Blocks: " << gridsize << endl;


	//--------------------------ITERATION LOOP-----------------------------
	cout << "Running Simulation...\n";

	time_t start = seconds();

	for (it = 0; it < ITERATIONS; it++)
	{
	
		//--------------------------CILIA BEAT DEFINITION-------------------------

		
			for (m = 0; m < c_num; m++)
			{
				if (it + m*p_step == T) phase = T;
				else phase = (it + m*p_step) % T;

				offset = 1.*(m - (c_num - 1) / 2.)*c_space;


				define_filament << <10, 1000 >> > (m, T, phase, offset, d_boundary, d_lasts);

				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) { fprintf(stderr, "define_filament failed: %s\n", hipGetErrorString(cudaStatus)); }

				define_boundary << <1, 100 >> > (m, c_num, d_boundary, d_b_points);

				cudaStatus = hipGetLastError();
				if (cudaStatus != hipSuccess) { fprintf(stderr, "define_boundary failed: %s\n", hipGetErrorString(cudaStatus)); }


				cudaStatus = hipMemcpy(b_points, d_b_points, 5 * Np * sizeof(double), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy of b_points failed!\n"); }

			}

		
			for (j = 0; j < c_num*LENGTH; j++)
			{
				k = j;

				s[2 * k + 0] = n*LENGTH / 2.*c_num + (LENGTH / 2.*c_num) / 2. + b_points[5 * j + 0];

				if (s[2 * k + 0] < 0) s[2 * k + 0] += XDIM;
				else if (s[2 * k + 0] > XDIM) s[2 * k + 0] -= XDIM;

				s[2 * k + 1] = b_points[5 * j + 1];

				if (it == 0)
				{
					u_s[2 * k + 0] = 0.;
					u_s[2 * k + 1] = 0.;
				}
				else
				{
					u_s[2 * k + 0] = b_points[5 * j + 2];
					u_s[2 * k + 1] = b_points[5 * j + 3];
				}

				E += u_s[2 * k + 0];

				epsilon[k] = 1;
			}
		
		
		for (m = 0; m < c_num; m++)
		{
			boundary_check(n*c_num + m, c_space, c_num, LENGTH, s, epsilon);
		}
		
		//---------------------------CILIUM COPY---------------------------------------- 

		{

			cudaStatus = hipMemcpy(d_epsilon, epsilon, Ns * sizeof(int), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of epsilon failed!\n");
			}

			cudaStatus = hipMemcpy(d_s, s, 2 * Ns * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of s failed!\n");
			}

			cudaStatus = hipMemcpy(d_u_s, u_s, 2 * Ns * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of u_s failed!\n");
			}

			cudaStatus = hipMemcpy(d_F_s, F_s, 2 * Ns * sizeof(double), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of F_s failed!\n");
			}
		}


		//---------------------------IMMERSED BOUNDARY LATTICE BOLTZMANN STEPS-------------------

		equilibrium << <gridsize, blocksize >> > (d_u, d_rho, d_f0, d_force, d_F, XDIM, YDIM, TAU);				//EQUILIBRIUM STEP

		{																										// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "equilibrium launch failed: %s\n", hipGetErrorString(cudaStatus));
			}
		}

		collision << <gridsize, blocksize >> > (d_f0, d_f, d_f1, d_F, TAU, TAU2, XDIM, YDIM, it);						//COLLISION STEP

		{																										// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "collision launch failed: %s\n", hipGetErrorString(cudaStatus));
			}
		}

		streaming << <gridsize, blocksize >> > (d_f1, d_f, XDIM, YDIM);												//STREAMING STEP

		{																											// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "collision launch failed: %s\n", hipGetErrorString(cudaStatus));
			}

		}

		macro << <gridsize, blocksize >> > (d_f, d_u, d_rho, XDIM, YDIM);											//MACRO STEP

		{
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "collision launch failed: %s\n", hipGetErrorString(cudaStatus));
			}
		}

		interpolate << <gridsize2, blocksize2 >> > (d_rho, d_u, Ns, d_u_s, d_F_s, d_s, XDIM);											//IB INTERPOLATION STEP

		{
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "interpolate launch failed: %s\n", hipGetErrorString(cudaStatus));
			}
		}

		spread << <gridsize, blocksize >> > (d_rho, d_u, d_f, Ns, d_u_s, d_F_s, d_force, d_s, XDIM, d_Q, d_epsilon);	//IB SPREADING STEP

		{
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "spread launch failed: %s\n", hipGetErrorString(cudaStatus));
				
				//cout << it << endl;
				//system("pause");
				return 1;
			}

			cudaStatus = hipMemcpy(rho, d_rho, size * sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of rho failed!\n");
			}

			cudaStatus = hipMemcpy(u, d_u, 2 * size * sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of u failed!\n");
			}

			cudaStatus = hipMemcpy(&Q, d_Q, sizeof(double), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy of u failed!\n");
			}
		}

		//----------------------------DATA OUTPUT------------------------------


		double t_scale = 1000.*dt*t_0;					//milliseconds
		double x_scale = 1000000. * dx*l_0;				//microns
		double s_scale = 1000.*x_scale / t_scale;		//millimetres per second


		if (it % INTERVAL == 0)
		{
			if (BigData)
			{
				outfile = raw_data + to_string(it) + "-fluid.dat";

				fsA.open(outfile.c_str());

				for (j = 0; j < XDIM*YDIM; j++)
				{
					int x = j%XDIM;
					int y = (j - j%XDIM) / XDIM;

					double ab = sqrt(u[2 * j + 0] * u[2 * j + 0] + u[2 * j + 1] * u[2 * j + 1]);

					fsA << x*x_scale << "\t" << y*x_scale << "\t" << u[2 * j + 0]*s_scale << "\t" << u[2 * j + 1]*s_scale << "\t" << ab*s_scale << "\t" << rho[j] << endl;


					if (x == XDIM - 1) fsA << endl;
				}

				fsA.close();

				outfile = cilia_data + to_string(it) + "-cilia.dat";

				fsA.open(outfile.c_str());

				for (k = 0; k < Ns; k++)
				{
					fsA << s[2 * k + 0] << "\t" << s[2 * k + 1] << "\t" << u_s[2 * k + 0] << "\t" << u_s[2 * k + 1] << "\t" << epsilon[k] << "\n"; //LOOP FOR Np
					if (k % 100 == 99 || s[2 * k + 0] > XDIM - 1 || s[2 * k + 0] < 1) fsA << "\n";
				}

				fsA.close();

			}
			
			fsB.open(flux.c_str(), ofstream::app);

			fsB << it*1000.*dt*t_0 << "\t" << Q*1000000. * dx*l_0*1000000. * dx*l_0 << "\t" << E*1000000. * dx*l_0*1000000. * dx*l_0 << endl;

			fsB.close();
		}

		if (it == INTERVAL)
		{
			time_t cycle = seconds();

			p_runtime = (cycle - start)*(ITERATIONS / INTERVAL);

			time_t p_end = rawtime + p_runtime;

			timeinfo = localtime(&p_end);

			//int hours(0), mins(0);
			//time_t secs(0.);

			//if (p_runtime >= 3600) hours = nearbyint(p_runtime / 3600 - 0.5);
			//if (p_runtime >= 60) mins = nearbyint((p_runtime - hours * 3600) / 60 - 0.5);
			//secs = p_runtime - hours * 3600 - mins * 60;

			//cout << "\nProjected runtime: ";
			//if (hours < 10) cout << 0;
			//cout << hours << ":";
			//if (mins < 10) cout << 0;
			//cout << mins << ":";
			//if (secs < 10) cout << 0;
			//cout << fixed << setprecision(2) << secs;

			cout << "\nCompletion time: " << asctime(timeinfo) << endl;

			fsC << "\nCompletion time: " << asctime(timeinfo) << endl;

			fsC.close();
		}

	}

	fsB.open(flux.c_str(), ofstream::app);

	fsB << it*1000.*dt*t_0 << "\t" << Q*1000000. * dx*l_0*1000000. * dx*l_0 << endl;

	fsB.close();
	
	double end = seconds();

	double runtime = end - start;

	int hours(0), mins(0);
	double secs(0.);

	if (runtime > 3600) hours = nearbyint(runtime / 3600 - 0.5);
	if (runtime > 60) mins = nearbyint((runtime - hours * 3600) / 60 - 0.5);
	secs = runtime - hours * 3600 - mins * 60;

	fsC.open(parameters.c_str(), ofstream::app);

	fsC << "\nTotal runtime: ";
	if (hours < 10) fsC << 0;
	fsC << hours << ":";
	if (mins < 10) fsC << 0;
	fsC << mins << ":";
	if (secs < 10) fsC << 0;
	fsC << secs << endl;
	fsC << "Net Q = " << Q << " Avg Q = " << Q / 1.*(it / T) << endl;

	fsC.close();

	hipDeviceReset();


	return 0;
}